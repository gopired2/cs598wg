#include "hip/hip_runtime.h"
/***
*	Ashutosh Dhar
*	Department of Electrical and Computer Engineeing
*	University of Illinois, Urbana-Champaign
*	
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include "support.h"

#define THREADS_PER_SM 2048
#define BLOCKS_PER_SM 32

using namespace std;

// Vector addition Kernel
// Device kernel

__global__ void bandwidthTest(const float *A, float *C, int numElements, int n_thread)
{
        long unsigned tid = blockDim.x * blockIdx.x + (threadIdx.x);
	int i;

        if (tid < numElements)
        {
			for(i=0; i<n_thread; i++)
	                        C[tid + (i * THREADS_PER_SM * BLOCKS_PER_SM)] = 2*A[tid + (i * THREADS_PER_SM * BLOCKS_PER_SM)];
        }

}

int main(int argc, char **argv)
{

	long unsigned n_elements_thread;
	long unsigned n_threads;
	long unsigned n_elements;
        long unsigned n_threads_per_block = THREADS_PER_SM/BLOCKS_PER_SM; //2048 threads per SM, 16 blocks per SM
	unsigned n_test = 5;
	
	Timer t_timer;

	if(argc > 1)
	{
		n_elements_thread = atoi(argv[1]);
	}
	else
	{
		n_elements_thread = 1;
	}
	
	if(argc > 2)
	{
		n_test = atoi(argv[2]);
	}

	n_elements = n_elements_thread * THREADS_PER_SM * BLOCKS_PER_SM;
	n_threads = n_elements / n_elements_thread;
	
	printf("Num elements in generated vector: %lu\n", n_elements);
	printf("Threads Per Block: %lu\n", n_threads_per_block);	
	
    	//Error Flag returned by cuda
    	hipError_t errorFlag = hipSuccess;
//	hipError_t err = hipSuccess;

    	//Set element sizes and allocate data sizes
    	int numElements = n_elements;
    	size_t sizeVector = (numElements * sizeof(float));

    	// Allocate the host input vector A
    	float *h_A = (float *)malloc(sizeVector);
    	// Allocate the host input vector B
//    	float *h_B = (float *)malloc(sizeVector);
    	// Allocate the host output vector C
    	float *h_C = (float *)malloc(sizeVector);

// From NVIDIA SDK, randomized initialization of data

    	// Verify that allocations succeeded
    	if (h_A == NULL || h_C == NULL)
    	{
        	fprintf(stderr, "Failed to allocate host vectors!\n");
        	exit(EXIT_FAILURE);
    	}

    	// Initialize the host input vectors
    	for (int i = 0; i < numElements; ++i)
    	{
        	h_A[i] = rand()/(float)RAND_MAX;
//        	h_B[i] = rand()/(float)RAND_MAX;
    	}

    	// Allocate the device input vector A
    	float *d_A = NULL;
    	errorFlag = hipMalloc((void **)&d_A, sizeVector);

    	if (errorFlag != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(errorFlag));
        	exit(EXIT_FAILURE);
    	}

/*
    	// Allocate the device input vector B
    	float *d_B = NULL;
    	errorFlag = hipMalloc((void **)&d_B, sizeVector);

    	if (errorFlag != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(errorFlag));
        	exit(EXIT_FAILURE);
    	}
*/
    	// Allocate the device output vector C
    	float *d_C = NULL;
    	errorFlag = hipMalloc((void **)&d_C, sizeVector);

    	if (errorFlag != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(errorFlag));
        	exit(EXIT_FAILURE);
    	}

	    // Copy the host input vectors A and B in host memory to the device input vectors in device memory

	    errorFlag = hipMemcpy(d_A, h_A, sizeVector, hipMemcpyHostToDevice);

	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }


/*	    errorFlag = hipMemcpy(d_B, h_B, sizeVector, hipMemcpyHostToDevice);

	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }

*/
	    // Launch the Vector Add CUDA Kernel
	    int threadsPerBlock = n_threads_per_block;
	    int blocksPerGrid = n_threads/n_threads_per_block;

	    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		
	    double t_time = 10000000;

	    for(int k=0; k<n_test; k++)
	    {
	    	startTime(&t_timer);
	    	bandwidthTest<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements,n_elements_thread);
		hipDeviceSynchronize();
	    	stopTime(&t_timer);
	    	errorFlag = hipGetLastError();
	 
	    	if (errorFlag != hipSuccess)
	    	{
			fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(errorFlag));
			exit(EXIT_FAILURE);
	    	}
		
		if(elapsedTime(t_timer) < t_time)
		{
			t_time = elapsedTime(t_timer);
		}

            }

	    // Copy the device result vector in device memory to the host result vector
	    // in host memory.
	    printf("Copy output data from the CUDA device to the host memory\n");
	    errorFlag = hipMemcpy(h_C, d_C, sizeVector, hipMemcpyDeviceToHost);

	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }


	    // Free device global memory
	    errorFlag = hipFree(d_A);

/*
	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }
	    errorFlag = hipFree(d_B);
*/
	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }
	    errorFlag = hipFree(d_C);

	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }

	    // Free host memory
	    free(h_A);
//	    free(h_B);
	    free(h_C);

	    // Reset the device and exit
	    errorFlag = hipDeviceReset();

	    if (errorFlag != hipSuccess)
	    {
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(errorFlag));
		exit(EXIT_FAILURE);
	    }

	    double bw = 2.0*(1.0*n_elements/(1000.00*1000.00))/(t_time);

	    printf("Num Elements: %lu Time: %e Bandwidth(MB/s): %e\n",n_elements,t_time,bw);
	    return 0;
}
