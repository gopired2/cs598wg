#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <math.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    int THREADS_PER_BLOCK = 128;
    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    }  else if(argc == 3) {
        n = atoi(argv[1]);
	THREADS_PER_BLOCK = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    float* A_d ; 
    int success = hipMalloc( (void**) &A_d, sizeof(float)*n) ; 

    float* B_d ; 
    success = hipMalloc( (void**) &B_d, sizeof(float)*n) ; 


    float* C_d ; 
    success = hipMalloc( (void**) &C_d, sizeof(float)*n) ; 

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    success = hipMemcpy (A_d,A_h,sizeof(float)*n, hipMemcpyHostToDevice);

    success = hipMemcpy (B_d,B_h,sizeof(float)*n, hipMemcpyHostToDevice);


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

/*
    int numBlocks = ceil((n-1)/256) + 1;
    printf("NumBlocks %f\n",numBlocks) ;

    vecAddKernel<<<numBlocks,256>>>(A_d,B_d,C_d,n) ;
*/
	dim3 DimGrid(ceil(1.0*n/THREADS_PER_BLOCK),1,1) ;
	dim3 DimBlock(THREADS_PER_BLOCK,1,1) ; 

   printf("DimGrid: %dX%dX%d \nDimBlock: %dX%dX%d\n", DimGrid.x, DimGrid.y, DimGrid.z, DimBlock.x, DimBlock.y, DimBlock.z ); fflush(stdout);

    vecAddKernel<<<DimGrid,DimBlock>>>(A_d,B_d,C_d,n) ;
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    success = hipMemcpy (C_h,C_d,sizeof(float)*n, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;

}

