/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"
#include "timer.cu"
// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
    return;
}

int main (int argc, char *argv[])
{

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1500;
        matBcol = 1300;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
                "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
                "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
                "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
                "\n");
        exit(0);
    }

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
            matBrow, matBcol, matArow, matBcol);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);


    cuda_ret = hipMalloc ((void**) &A_d, sizeof(float)* A_sz  ) ;
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");


    cuda_ret = hipMalloc ((void**) &B_d, sizeof(float)* B_sz  ) ;
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc ((void**) &C_d, sizeof(float)* C_sz  ) ;
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);



    cuda_ret = hipMemcpy (A_d ,A_h ,sizeof(float) * A_sz , hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Memcpy to device failed");

    cuda_ret = hipMemcpy (B_d ,B_h ,sizeof(float) * B_sz , hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Memcpy to device failed");


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface ---------------------------
    kernelTimer(matArow, matBcol, matBrow, A_d, B_d, C_d);
    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);


    cuda_ret = hipMemcpy (C_h ,C_d ,sizeof(float) * C_sz , hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Memcpy to device failed");


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    //verify(A_h, B_h, C_h, matArow, matAcol, matBcol);

/*
    //properties
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }
*/
    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);


    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;

}

